#include "hip/hip_runtime.h"
 /*
 * PFNipsLayersKernels.cpp
 * Copyright (C) 2020 M. Rosario Cervellere <rosario.cervellere@gmail.com>
 *
 * Distributed under terms of the MIT license.
 */

#include "PFNipsLayersKernels.h"
#include <stdio.h>
#include <math.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>



// -------------------------------------------------------
// Device Functions
// -------------------------------------------------------


/**********************************************************
   * Laplacian of non-uniform mobility field 
   * for Cahn-Hilliard Euler update
   ********************************************************/

__device__ double laplacianNonUniformMob_NIPS(double *f, double *Mob,int gid, int x, int y, int z,
                                         int nx, int ny, int nz, double h, bool bX, bool bY, bool bZ)
{
	// get id of neighbors for no-flux and PBCs
   int xlid,xrid,ylid,yrid,zlid,zrid;
   // -----------------------------------
   // X-Direction Boundaries
   // -----------------------------------
	if (bX) {
		// PBCs (x-dir.)
		if(x == 0) xlid = nx*ny*z + nx*y + nx-1;
		else xlid = nx*ny*z + nx*y + x-1;
		if(x == nx-1) xrid = nx*ny*z + nx*y + 0;
		else xrid = nx*ny*z + nx*y + x+1;
	}
	else {
	 	// no-flux BC (x-dir.)
		if (x == 0) xlid = nx*ny*z + nx*y + x;
		else xlid = nx*ny*z + nx*y + x-1;
		if (x == nx-1) xrid = nx*ny*z + nx*y + x;
		else xrid = nx*ny*z + nx*y + x+1;
   }
   // -----------------------------------
   // Y-Direction Boundaries
   // -----------------------------------
	if (bY) {
		// PBC Apply
	   if(y == 0) ylid = nx*ny*z + nx*(ny-1) + x;
    	else ylid = nx*ny*z + nx*(y-1) + x;
    	if(y == ny-1) yrid = nx*ny*z + nx*0 + x;
    	else yrid = nx*ny*z + nx*(y+1) + x;
   }
   else {
   	// no-flux BC (y-dir.)
      if(y == 0) ylid = nx*ny*z + nx*y + x;
    	else ylid = nx*ny*z + nx*(y-1) + x;
    	if(y == ny-1) yrid = nx*ny*z + nx*y + x;
    	else yrid = nx*ny*z + nx*(y+1) + x;
	}
   // -----------------------------------
   // Z-Direction Boundaries
   // -----------------------------------
	if (bZ) {
		// PBC Apply (z-dir.)
   	if(z == 0) zlid = nx*ny*(nz-1) + nx*y + x;
    	else zlid = nx*ny*(z-1) + nx*y + x;
    	if(z == nz-1) zrid = nx*ny*0 + nx*y + x;
    	else zrid = nx*ny*(z+1) + nx*y + x;
   }
	else {
		// no-flux BC (z-dir.)
		if(z == 0) zlid = nx*ny*z + nx*y + x;
    	else zlid = nx*ny*(z-1) + nx*y + x;
    	if(z == nz-1) zrid = nx*ny*z + nx*y + x;
    	else zrid = nx*ny*(z+1) + nx*y + x;
	}

    // ------------------------------------------
    // begin laplacian
	// ------------------------------------------
	
    // get values of neighbors for mobility
    double mobXl = Mob[xlid];
    double mobXr = Mob[xrid];
    double mobYl = Mob[ylid];
    double mobYr = Mob[yrid];
    double mobZl = Mob[zlid];
    double mobZr = Mob[zrid];
    // get values of neighbors for mu
    double xl = f[xlid];
    double xr = f[xrid];
    double yl = f[ylid];
    double yr = f[yrid];
    double zl = f[zlid];
    double zr = f[zrid];
    // get value of current points
    double bo = Mob[gid];
    double fo = f[gid];
    // begin laplacian
    double bx1 = 2.0/(1.0/mobXl + 1.0/bo);
    double bx2 = 2.0/(1.0/mobXr + 1.0/bo);
    double by1 = 2.0/(1.0/mobYl + 1.0/bo);
    double by2 = 2.0/(1.0/mobYr + 1.0/bo);
    double bz1 = 2.0/(1.0/mobZl + 1.0/bo);
    double bz2 = 2.0/(1.0/mobZr + 1.0/bo);
    double lapx = (xl*bx1 + xr*bx2 - (bx1+bx2)*fo)/(h*h); 
    double lapy = (yl*by1 + yr*by2 - (by1+by2)*fo)/(h*h);
    double lapz = (zl*bz1 + zr*bz2 - (bz1+bz2)*fo)/(h*h);
    double lapNonUniform = lapx + lapy + lapz;
    return lapNonUniform;
}   
   

/*********************************************************
   * Compute Laplacian with user specified 
   * boundary conditions (UpdateBoundaries)
   ******************************************************/
	
__device__ double laplacianUpdateBoundaries_NIPS(double* f,int gid, int x, int y, int z, 
								            int nx, int ny, int nz, double h, 
								            bool bX, bool bY, bool bZ)
{
    // get id of neighbors with periodic boundary conditions
    // and no-flux conditions
    int xlid,xrid,ylid,yrid,zlid,zrid;
    // -----------------------------------
    // X-Direction Boundaries
    // -----------------------------------
    if (bX) {
        // PBCs (x-dir.)
        if(x == 0) xlid = nx*ny*z + nx*y + nx-1;
        else xlid = nx*ny*z + nx*y + x-1;
        if(x == nx-1) xrid = nx*ny*z + nx*y + 0;
        else xrid = nx*ny*z + nx*y + x+1;
    }
    else {
        // no-flux BC (x-dir.)
		if (x == 0) xlid = nx*ny*z + nx*y + x;
		else xlid = nx*ny*z + nx*y + x-1;
		if (x == nx-1) xrid = nx*ny*z + nx*y + x;
		else xrid = nx*ny*z + nx*y + x+1;
    }
    // -----------------------------------
    // Y-Direction Boundaries
    // -----------------------------------
	if (bY) {
        // PBC Apply
        if(y == 0) ylid = nx*ny*z + nx*(ny-1) + x;
    	else ylid = nx*ny*z + nx*(y-1) + x;
    	if(y == ny-1) yrid = nx*ny*z + nx*0 + x;
    	else yrid = nx*ny*z + nx*(y+1) + x;
    }
    else {
   	// no-flux BC (y-dir.)
        if(y == 0) ylid = nx*ny*z + nx*y + x;
    	else ylid = nx*ny*z + nx*(y-1) + x;
    	if(y == ny-1) yrid = nx*ny*z + nx*y + x;
    	else yrid = nx*ny*z + nx*(y+1) + x;
    }
    // -----------------------------------
    // Z-Direction Boundaries
    // -----------------------------------
	if (bZ) {
		// PBC Apply (z-dir.)
   	if(z == 0) zlid = nx*ny*(nz-1) + nx*y + x;
    	else zlid = nx*ny*(z-1) + nx*y + x;
    	if(z == nz-1) zrid = nx*ny*0 + nx*y + x;
    	else zrid = nx*ny*(z+1) + nx*y + x;
    }
	else {
		// no-flux BC (z-dir.)
		if(z == 0) zlid = nx*ny*z + nx*y + x;
    	else zlid = nx*ny*(z-1) + nx*y + x;
    	if(z == nz-1) zrid = nx*ny*z + nx*y + x;
    	else zrid = nx*ny*(z+1) + nx*y + x;
	}
    // get values of neighbors
    double xl = f[xlid];
    double xr = f[xrid];
    double yl = f[ylid];
    double yr = f[yrid];
    double zl = f[zlid];
    double zr = f[zrid];
    double lap = (xl+xr+yl+yr+zl+zr-6.0*f[gid])/(h*h);
    return lap;
}


/*************************************************************
  * compute chi with linear weighted average
  ***********************************************************/

__device__ double chiDiffuse_NIPS(double locWater, double chiPS, double chiPN)
{
    double chi = chiPN*locWater + chiPS*(1.0-locWater);
	return chi;
}


/*************************************************************
	* Compute the chemical potential using the 1st derivative
	* of the  binary Flory-Huggins free energy of mixing with
	* respect to c
	*
	* F = c*log(c)/N + (1-c)*log(1-c) + chi*c*(1-c)
	*
	*
	* dF/dc = (log(c) + 1)/N - log(1 - c) - 1.0 
	*         + chi*(1 - 2*c)
	*
	***********************************************************/

/*__device__ double freeEnergyBiFH_NIPS(double cc, double chi, double N, double lap_c, double kap, double A)
{
   double c_fh = 0.0;
   if (cc < 0.0) c_fh = 0.0001;
   else if (cc > 1.0) c_fh = 0.999;
   else c_fh = cc;
   double FH = (log(c_fh) + 1.0)/N - log(1.0-c_fh) - 1.0 + chi*(1.0-2.0*c_fh) - kap*lap_c;
   if (cc <= 0.0) FH = -1.5*A*sqrt(-cc) - kap*lap_c;   
   return FH;
}*/


__device__ double freeEnergyTernaryFH_NIPS(double cc, double cc1, double chi, double N, double lap_c, double kap, double A)
{
    double cc_fh = 0.0;
    double cc1_fh = 0.0;
    double chi_pp = 0.01;
    if (cc <= 0.0) cc_fh = 0.0001;
    else if (cc >= 1.0) cc_fh = 0.999;
    else cc_fh = cc;
    if (cc1 <= 0.0) cc1_fh = 0.0001;
    else if (cc1 >= 1.0) cc1_fh = 0.999;
    else cc1_fh = cc1;
    double n_fh = 1.0 - cc_fh - cc1_fh;
    // double FH = (chi*N*(cc1_fh + n_fh) + 2*log(cc_fh)+ 2)/(2*N) - kap*lap_c; // this assumes chi = chi_12 = chi_13 = chi_23
    // above equation not right...
    // 1st derivative from FH from Tree et. al 2019
    double FH = ((chi_pp*N*cc_fh) + (chi*N*n_fh) + 2*log(cc1_fh) + 2)/(2*N);
    // subtract kap*lap_c for CH
    FH -= kap*lap_c;
    // if our values go over 1 or less than 0, push back toward [0,1]
    if (cc < 0.0) FH = -1.5*A*sqrt(-cc) - kap*lap_c;  
    if (cc > 1.0) FH = 1.5*A*sqrt(cc - 1.0) - kap*lap_c;
    return FH;
}

/*************************************************************
  * Compute second derivative of FH with respect to phi
  ***********************************************************/
  
/*__device__ double d2dc2_FH_NIPS(double cc, double N)
{
   double c2_fh = 0.0;
   if (cc < 0.0) c2_fh = 0.0001;
   else if (cc > 1.0) c2_fh = 0.999;
   else c2_fh = cc;
   double FH_2 = 0.5 * (1.0/(N*c2_fh) + 1.0/(1.0-c2_fh));
   return FH_2;	
}*/

/*************************************************************
  * Compute diffusion coefficient via phillies eq.
  ***********************************************************/

/*__device__ double philliesDiffusion_NIPS(double cc, double gamma, double nu, 
								    double D0, double Mweight, double Mvolume)
{
	double cc_d = 1.0;
	double rho = Mweight/Mvolume;
	if (cc >= 1.0) cc_d = 1.0 * rho; // convert phi to g/L	
	else if (cc < 0.0) cc_d = 0.0001 * rho; // convert phi to g/L 
	else cc_d = cc * rho; // convert phi to g/L
	double Dp = D0 * exp(-gamma * pow(cc_d,nu));
	return Dp;
}*/


// -------------------------------------------------------
// Device Kernels for Testing
// -------------------------------------------------------


/****************************************************************
  * Kernels for unit testing the laplacian devices 
  ***************************************************************/

__global__ void testLap_NIPS(double* f, int nx, int ny, int nz, double h, bool bX, bool bY, bool bZ)
{
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        f[gid] = laplacianUpdateBoundaries_NIPS(f,gid,idx,idy,idz,nx,ny,nz,h,bX,bY,bZ);
    }
}

__global__ void testLapNonUniformMob_NIPS(double* f, double *Mob, int nx, int ny, int nz, double h, bool bX, bool bY, bool bZ)
{
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        f[gid] = laplacianNonUniformMob_NIPS(f,Mob,gid,idx,idy,idz,nx,ny,nz,h,bX,bY,bZ);
    }
}




// -------------------------------------------------------
// Device Kernels for Simulation
// -------------------------------------------------------


/*********************************************************
  * Compute the laplacian of the concentration array c and w
  * and store it in the device array df and wdf
  *******************************************************/

__global__ void calculateLapBoundaries_NIPS(double* c,double* df, int nx, int ny, int nz, 
													double h, bool bX, bool bY, bool bZ)
{
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        df[gid] = laplacianUpdateBoundaries_NIPS(c,gid,idx,idy,idz,nx,ny,nz,h,bX,bY,bZ);
    }
}




/*********************************************************
  * Computes the chemical potential of a concentration
  * order parameter and stores it in the df_d array.
  *******************************************************/


__global__ void calculateChemPotFH_NIPS(double* c,double* c1,double* w,double* df,/*double*df1,*/ double kap, double A, double chiPS, double chiPN, double N, int nx, int ny, int nz, int current_step, double dt)
{
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        double cc = c[gid];
        double cc1 = c1[gid];
        double ww = w[gid];
        double lap_c = df[gid];
        // double lap_c1 = df1[gid];
        // compute interaction parameter
        double chi = chiDiffuse_NIPS(ww,chiPS,chiPN);
        // compute chemical potential
        // df[gid] = freeEnergyBiFH_NIPS(cc,chi,N,lap_c,kap,A);
        df[gid] = freeEnergyTernaryFH_NIPS(cc,cc1,chi,N,lap_c,kap,A);
        // df1[gid] = freeEnergyTernaryFH_NIPS(cc1,cc,chi,N,lap_c1,kap,A);
    }
}


/*********************************************************
  * Computes the mobility of a concentration order
  * parameter and stores it in the Mob_d array.
  *******************************************************/
  
__global__ void calculateMobility_NIPS(double* c,double* Mob, double M,double mobReSize, int nx, int ny, int nz,
											 double phiCutoff, double N,
        									 double gamma, double nu, double D0, double Mweight, double Mvolume, double Tcast)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        M = 1.0;
        int gid = nx*ny*idz + nx*idy + idx;
        double cc = c[gid];
        //double FH2 = d2dc2_FH_NIPS(cc,N);
        //double D_phil = philliesDiffusion_NIPS(cc,gamma,nu,D0,Mweight,Mvolume);
        //double Dtemp = D0*Tcast/273.15;
        //M = Dtemp*D_phil/FH2;
        //if (M > 1.0) M = 1.0;     // making mobility max = 1
        //else if (M < 0.0) M = 0.001; // mobility min = 0.001
        // Using phiCutoff as vitrification
        //if (cc > phiCutoff) { 
        //    M *= 1e-6;
        //}
        // resize mobility to be similar to experiments
        //M *= mobReSize;
       // Mob[gid] = M;		  
    }
}

/************************************************************************************
  * Computes the non-uniform mobility and chemical potential laplacian, multiplies 
  * it by the time step to get the RHS of the CH equation, then uses this RHS value 
  * to perform an Euler update of the concentration in time.
  ***********************************************************************************/

__global__ void lapChemPotAndUpdateBoundaries_NIPS(double* c, double* c1, double* df, double* df1, double* Mob,/*double* nonUniformLap,*/ double M, double M1, double dt, int nx, int ny, int nz, double h,bool bX, bool bY, bool bZ)
{
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        // compute chemical potential laplacain with non-uniform mobility
        // and user defined boundaries (no-flux or PBCs)
        //nonUniformLap[gid] = laplacianNonUniformMob_NIPS(df,Mob,gid,idx,idy,idz,nx,ny,nz,h,bX,bY,bZ);
        //c[gid] += nonUniformLap[gid]*dt;
        
        // calculate non-uniform laplacian without nonUniform array/field (save memory)
        // do euler update
        // double nonUniformLap_c = laplacianNonUniformMob_NIPS(df,Mob,gid,idx,idy,idz,nx,ny,nz,h,bX,bY,bZ);
        // c[gid] += nonUniformLap_c*dt;
        
        // compute laplacian of chemical potential and update with constant mobility
        // compute laplacian and do euler update
        double cc = c[gid];
        double cc1 = c1[gid];
        if (cc + cc1 >= 0.75) {M1 = 0; M = 0;}
        //if (cc1 > 0.75) M = 0;
        double lap_c = laplacianUpdateBoundaries_NIPS(df,gid,idx,idy,idz,nx,ny,nz,h,bX,bY,bZ);
        double lap_c1 = laplacianUpdateBoundaries_NIPS(df1,gid,idx,idy,idz,nx,ny,nz,h,bX,bY,bZ); // commented out to save memory
        c[gid] += M1*lap_c1*dt + M*lap_c*dt;
        c1[gid] += M1*lap_c*dt + M*lap_c1*dt; // commented out to save memory
    } 
}



/*__global__ void calculate_muNS_NIPS(double*w, double*c, double* muNS, double* Mob, double Dw, double water_CB, double gamma, double nu, double Mweight, double Mvolume, int nx, int ny, int nz)
{
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        
        // calculate mu for NonSolvent NS diffusion
        // make x = 0 coagulation bath composition
        if (idx == 0) w[gid] = water_CB;
        double ww = w[gid];
        // check that polymer < 1.0 and greater than 0.0
        double cc = c[gid];
        if (cc < 0.0) cc = 0.0;
        else if (cc > 1.0) cc = 1.0;
        
        // assign muNS for calculating laplacian
        muNS[gid] =  ww;
        
        double D_NS_phil = philliesDiffusion_NIPS(cc,gamma,nu,Dw,Mweight,Mvolume);
        Mob[gid] = D_NS_phil;
        if (Mob[gid] < 0.0) Mob[gid] = 0.0;
    }
    
}*/



__global__ void calculateLapBoundaries_muNS_NIPS(double* df, double* muNS, int nx, int ny, int nz, double h, bool bX, bool bY, bool bZ)
{
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        df[gid] = laplacianUpdateBoundaries_NIPS(muNS,gid,idx,idy,idz,nx,ny,nz,h,bX,bY,bZ);
    }
}

/*__global__ void calculateNonUniformLapBoundaries_muNS_NIPS(double* muNS, double* Mob,double* nonUniformLap, int nx, int ny, int nz, double h, bool bX, bool bY, bool bZ)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        nonUniformLap[gid] = laplacianNonUniformMob_NIPS(muNS,Mob,gid,idx,idy,idz,nx,ny,nz,h,bX,bY,bZ);
    }
}*/

__global__ void calculate_water_diffusion(double*w,double*c,double*c1,double*Mob,double Dw,double Dw1,double water_CB,int nx, int ny, int nz)
{
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        double cc = c[gid];
        double cc1 = c1[gid];
        double D = Dw*(cc) + Dw1*(cc1);
        Mob[gid] = D;
    }
}

__global__ void update_water_NIPS(double* w,double* df, double* Mob, /*double* nonUniformLap,*/ double dt, int nx, int ny, int nz, double h, bool bX, bool bY, bool bZ)
{
    // here we're re-using the Mob array for Dw_nonUniform
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        
        // removing nonUniformLap memory
        // nonUniformLap_w = laplacianNonUniformMob_NIPS(df,Mob,gid,idx,idy,idz,nx,ny,nz,h,bX,bY,bZ);
        // w[gid] += nonUniformLap_w*dt;
        
        // with nonUniformLap memory
        // w[gid] += nonUniformLap[gid]*dt;
        // check first layer...
        if (idx == 0) w[gid] = 1.0;
        else w[gid] += 10.0*df[gid]*dt;
    }
}


/**********************************************************************
  * initialize cuRAND for thermal fluctuations of polymerconcentration
  *********************************************************************/
__global__ void init_cuRAND_NIPS(unsigned long seed,hiprandState *state,int nx,int ny,int nz)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        hiprand_init(seed,gid,0,&state[gid]);
    }
}


/************************************************************
  * Add random fluctuations for non-trivial solution (cuRand)
  ***********************************************************/
__global__ void addNoise_NIPS(double *c,int nx, int ny, int nz, double dt, int current_step, 
                         double water_CB,double phiCutoff,hiprandState *state)
{
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        double noise = hiprand_uniform_double(&state[gid]);
        double cc = c[gid];
        double noiseScale = 1.0;
        // add random fluctuations with euler update
        if (cc > phiCutoff) noise = 0.5; // no fluctuations for phi < 0
        else if (cc <= 0.0) noise = 0.5;  // no fluctuations for phi > phiCutoff
        c[gid] += 0.1*(noise-0.5)*dt*noiseScale;
    }
}


/*********************************************************
  * Copies the contents of c into cpyBuffer so the c data
  * can be asynchronously transfered from the device to
  * the host.
  *******************************************************/

__global__ void populateCopyBuffer_NIPS(double* c,double* cpyBuff, int nx, int ny, int nz)
{
    // get unique thread id
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int idz = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx<nx && idy<ny && idz<nz)
    {
        int gid = nx*ny*idz + nx*idy + idx;
        // copy the contents of c to cpyBuff
        cpyBuff[gid] = c[gid];
    }
}
